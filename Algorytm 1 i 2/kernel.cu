
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <fstream>

using namespace std;


__global__ void chaotic(double *X, double *R, int y)
{
    int index = threadIdx.x + blockIdx.x * 1024;
    
    double sum = 0.0;
    double temp[48];
    
    if (X[index] >= 0.5)
    {
        for (int i = 1; i < y; i++)
        {
           // X[index] = X[index] * R[index] * (1 - X[index]);
        }
        R[index] = 3.99;
    }
    else
    {
        for (int i = 1; i < y; i++)
        {
            X[index] = X[index] * R[index] * (1 - X[index]);
            temp[i] = 3.86 + (X[index] * 0.14);
            sum = sum + temp[i];
            
        }
        R[index] = sum / y;
    }
}

__global__ void postprocess(double* X, double* R, unsigned int* v)
{
    int index = threadIdx.x + blockIdx.x * 1024;

    unsigned int M1 = 0, M2 = 0, M3 = 0, M4 = 0;
    
    if (blockIdx.x == 0)
    {
        unsigned long long* wsk;
        wsk = reinterpret_cast<long long unsigned*>(&X[index]);
        unsigned long long C1 = *wsk;
        wsk = reinterpret_cast<long long unsigned*>(&X[index + 33]);
        unsigned long long C2 = *wsk % 1024;

        unsigned int lsbmask = 0x80000000;

        for (int i = 0; i < 32; i++)
        {
            if ((lsbmask & C1) > 0)
            {
                M1 = M1 | lsbmask;
            }
            if ((lsbmask & C2) > 0)
            {
                M3 = M3 | lsbmask;
            }
            lsbmask = lsbmask >> 1;
        }

        lsbmask = 0x80000000;
        unsigned long long msbmask = 0x8000000000000000;

        for (int i = 0; i < 32; i++)
        {
            if ((msbmask & C1) > 0)
            {
                M2 = M2 | lsbmask;
            }
            if ((msbmask & C2) > 0)
            {
                M4 = M4 | lsbmask;
            }
            lsbmask = lsbmask >> 1;
            msbmask = msbmask >> 1;
        }

        v[index] = ((M1 + M4) ^ M3) + M2;
    }
}


int main(void)
{    
    srand(time(NULL));
    fstream plik;
    plik.open("plik.txt", ios::out);

    const int Beta = 1024;

    double *X = new double[Beta];
    double *R = new double[Beta];
    unsigned int *v = new unsigned int[Beta];
   

    double* dX = new double[Beta];
    double* dR = new double[Beta];
    unsigned int* dv = new unsigned int[Beta];


    int size = Beta * sizeof(double);
    int sizev = Beta * sizeof(unsigned int);

    hipMalloc((void**)&dX, size);
    hipMalloc((void**)&dR, size);
    hipMalloc((void**)&dv, sizev);

    X = (double *)malloc(size);
    R = (double *)malloc(size);
    v = (unsigned int*)malloc(sizev);

    for (int i = 0; i < Beta; i++)
    {
            X[i] = (rand() % 1000) / 1000.0;

            while (X[i] < 0.01)
            {
                X[i] = (rand() % 1000) / 1000.0;
            }

            R[i] = 3.86 + (X[i] * 0.14);
        // plik << X[i] << " " <<R[i]<< endl;
    }
    
    hipMemcpy(dX, X, size, hipMemcpyHostToDevice);
    hipMemcpy(dR, R, size, hipMemcpyHostToDevice);

    chaotic <<< 32, 1024 >>> (dX, dR, 48);

    chaotic <<< 32, 1024 >>> (dX, dR, 2);
    postprocess <<< 32, 1024 >>> (dX, dR, dv);

    hipMemcpy(X, dX, size, hipMemcpyDeviceToHost);
    hipMemcpy(R, dR, size, hipMemcpyDeviceToHost);
    hipMemcpy(v, dv, sizev, hipMemcpyDeviceToHost);


    for (int i = 0; i < Beta; i++)
    {
       plik << v[i] << endl;
        //plik << X[i] << " " << R[i] << endl;
    }

    return 0;
}
