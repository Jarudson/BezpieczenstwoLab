#include "hip/hip_runtime.h"



#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <fstream>

using namespace std;

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}



__global__ void chaotic(double* X, double* R, int y)
{
    int index = threadIdx.x + blockIdx.x * 512;

	if (X[index]==0){X[index] = R[index]/4;}
	R[index] = R[index]+0.0001*X[index]+0.002;
	if (R[index]>=4){R[index] = 3.9+0.0025*R[index];}
        for (int i = 1; i < y; i++)
        {
            X[index] = X[index] * R[index]* (1 - X[index]);
        }
}

__global__ void postprocess(double* X, unsigned int* v)
{
    int index = threadIdx.x + blockIdx.x * 512;

    unsigned int M1 = 0, M2 = 0, M3 = 0, M4 = 0;

    if (blockIdx.x == 0)
    {
        unsigned long long* wsk;
        wsk = reinterpret_cast<long long unsigned*>(&X[index]);
        unsigned long long C1 = *wsk;
        wsk = reinterpret_cast<long long unsigned*>(&X[index + 33]);
        unsigned long long C2 = *wsk % 1024;

        unsigned int lsbmask = 0x80000000;

        for (int i = 0; i < 32; i++)
        {
            if ((lsbmask & C1) > 0)
            {
                M1 = M1 | lsbmask;
            }
            if ((lsbmask & C2) > 0)
            {
                M3 = M3 | lsbmask;
            }
            lsbmask = lsbmask >> 1;
        }

        lsbmask = 0x80000000;
        unsigned long long msbmask = 0x8000000000000000;

        for (int i = 0; i < 32; i++)
        {
            if ((msbmask & C1) > 0)
            {
                M2 = M2 | lsbmask;
            }
            if ((msbmask & C2) > 0)
            {
                M4 = M4 | lsbmask;
            }
            lsbmask = lsbmask >> 1;
            msbmask = msbmask >> 1;
        }

        v[index] = ((M1 + M4) ^ M3) + M2;
    }
}


int main(void)
{
    srand(time(NULL));

    const int Beta = 512;

    double* X = new double[Beta];
    double* R = new double[Beta];
    unsigned int* v = new unsigned int[Beta];

    double* dX = new double[Beta];
    double* dR = new double[Beta];
    unsigned int* dv = new unsigned int[Beta];

    int* check = new int[Beta];

    int size = Beta * sizeof(double);
    int sizev = Beta * sizeof(unsigned int);

    gpuErrchk(hipMalloc((void**)&dX, size));
    gpuErrchk(hipMalloc((void**)&dR, size));
    gpuErrchk(hipMalloc((void**)&dv, sizev));

    X = (double*)malloc(size);
    R = (double*)malloc(size);
    v = (unsigned int*)malloc(sizev);

    for (int i = 0; i < Beta; i++)
    {
        check[i] = 0;
        v[i] = 0;

        X[i] = (rand() % 1000) / 1000.0;

        while (X[i] < 0.01)
        {
            X[i] = (rand() % 1000) / 1000.0;
        }

        R[i] = 3.86 + (X[i] * 0.14);
    }
	
    gpuErrchk(hipMemcpy(dX, X, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dR, R, size, hipMemcpyHostToDevice));
	
    chaotic KERNEL_ARGS2(32, 512) (dX, dR, 50);
    postprocess KERNEL_ARGS2(32, 512) (dX, dv);
	
    gpuErrchk(hipMemcpy(X, dX, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(v, dv, sizev, hipMemcpyDeviceToHost));

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
	
    fstream plik;
    plik.open("plik.csv", ios::out);

    for (int i = 0; i < Beta; i++)
    {
        plik << X[i] <<";"<< v[i] <<endl;
    }
	
    return 0;
}